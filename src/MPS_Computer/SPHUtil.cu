#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "SPHUtil.cuh"
#include "AdvectUtil.h"

#include <thrust/host_vector.h>
#include <thrust/extrema.h>

#define DEBUG_PRINT		0

void mps::kernel::sph::ComputeBoundaryParticleVolume_0(const mps::BoundaryParticleParam& boundaryParticle, const mps::MeshMaterialParam& material, const mps::SpatialHashParam& hash)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = boundaryParticle.GetSize();
	if (nSize == 0) return;

	ComputeBoundaryParticleVolume_0_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> >
		(boundaryParticle, material, hash);
	CUDA_CHECK(hipPeekAtLastError());
}
void mps::kernel::sph::ComputeBoundaryParticleVolume_1(
	const mps::BoundaryParticleParam& boundaryParticle, const mps::MeshMaterialParam& material, const mps::SpatialHashParam& hash,
	const mps::BoundaryParticleParam& refBoundaryParticle, const mps::MeshMaterialParam& refMaterial, const mps::SpatialHashParam& refHash)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = boundaryParticle.GetSize();
	if (nSize == 0) return;

	ComputeBoundaryParticleVolume_1_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> >
		(boundaryParticle, material, hash, refBoundaryParticle, refMaterial, refHash);
	CUDA_CHECK(hipPeekAtLastError());
}
void mps::kernel::sph::ComputeBoundaryParticleVolume_2(const mps::BoundaryParticleParam& boundaryParticle, const mps::MeshMaterialParam& material)
{
	constexpr auto nBlockSize = 1024u;

	const auto nSize = boundaryParticle.GetSize();
	if (nSize == 0) return;

	ComputeBoundaryParticleVolume_2_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> >
		(boundaryParticle, material);
	CUDA_CHECK(hipPeekAtLastError());
}

void mps::kernel::sph::ComputeDensity_0(const mps::SPHParam& sph, const mps::SPHMaterialParam& material, const mps::SpatialHashParam& hash)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	ComputeDensity_0_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> >
		(sph, material, hash);
	CUDA_CHECK(hipPeekAtLastError());
}
void mps::kernel::sph::ComputeDensity_1(
	const mps::SPHParam& sph, const mps::SPHMaterialParam& sphMaterial, const mps::SpatialHashParam& sphHash, 
	const mps::BoundaryParticleParam& boundaryParticle, const mps::MeshMaterialParam& boundaryParticleMaterial, const mps::SpatialHashParam& boundarhParticleHash)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	ComputeDensity_1_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> >
		(sph, sphMaterial, sphHash, boundaryParticle, boundaryParticleMaterial, boundarhParticleHash);
	CUDA_CHECK(hipPeekAtLastError());
}
void mps::kernel::sph::ComputeDensity_2(const mps::SPHParam& sph, const mps::SPHMaterialParam& material)
{
	constexpr auto nBlockSize = 1024u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	ComputeDensity_2_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> >
		(sph, material);
	CUDA_CHECK(hipPeekAtLastError());
}

void mps::kernel::sph::ComputeDFSPHFactor_0(const mps::SPHParam& sph, const mps::SPHMaterialParam& material, const mps::SpatialHashParam& hash)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	ComputeDFSPHFactor_0_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> >
		(sph, material, hash);
	CUDA_CHECK(hipPeekAtLastError());
}
void mps::kernel::sph::ComputeDFSPHFactor_1(
	const mps::SPHParam& sph, const mps::SPHMaterialParam& sphMaterial, const mps::SpatialHashParam& sphHash,
	const mps::BoundaryParticleParam& boundaryParticle, const mps::MeshMaterialParam& boundaryParticleMaterial, const mps::SpatialHashParam& boundarhParticleHash)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	ComputeDFSPHFactor_1_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> >
		(sph, sphMaterial, sphHash, boundaryParticle, boundaryParticleMaterial, boundarhParticleHash);
	CUDA_CHECK(hipPeekAtLastError());
}
void mps::kernel::sph::ComputeDFSPHFactor_2(const mps::SPHParam& sph, const mps::SPHMaterialParam& material)
{
	constexpr auto nBlockSize = 1024u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	ComputeDFSPHFactor_2_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> >
		(sph, material);
	CUDA_CHECK(hipPeekAtLastError());
}

void mps::kernel::sph::ComputeDensityDelta_0(const mps::SPHParam& sph, const mps::SPHMaterialParam& material, const mps::SpatialHashParam& hash)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	ComputeDensityDelta_0_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> >
		(sph, material, hash);
	CUDA_CHECK(hipPeekAtLastError());
}
void mps::kernel::sph::ComputeDensityDelta_1(const mps::SPHParam& sph, const mps::SPHMaterialParam& sphMaterial, const mps::SpatialHashParam& sphHash, const mps::BoundaryParticleParam& boundaryParticle, const mps::MeshMaterialParam& boundaryParticleMaterial, const mps::SpatialHashParam& boundarhParticleHash)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	ComputeDensityDelta_1_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> >
		(sph, sphMaterial, sphHash, boundaryParticle, boundaryParticleMaterial, boundarhParticleHash);
	CUDA_CHECK(hipPeekAtLastError());
}

void mps::kernel::sph::ApplyDFSPH_0(const mps::SPHParam& sph, const mps::SPHMaterialParam& material, const mps::SpatialHashParam& hash)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	ApplyDFSPH_0_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> >
		(sph, material, hash);
	CUDA_CHECK(hipPeekAtLastError());
}
void mps::kernel::sph::ApplyDFSPH_1(const mps::SPHParam& sph, const mps::SPHMaterialParam& sphMaterial, const mps::SpatialHashParam& sphHash, const mps::BoundaryParticleParam& boundaryParticle, const mps::MeshMaterialParam& boundaryParticleMaterial, const mps::SpatialHashParam& boundarhParticleHash)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	ApplyDFSPH_1_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> >
		(sph, sphMaterial, sphHash, boundaryParticle, boundaryParticleMaterial, boundarhParticleHash);
	CUDA_CHECK(hipPeekAtLastError());
}
void mps::kernel::sph::ApplyDFSPH_2(const mps::SPHParam& sph, const mps::SPHMaterialParam& material)
{
	constexpr auto nBlockSize = 1024u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	ApplyDFSPH_2_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> >
		(sph, material);
	CUDA_CHECK(hipPeekAtLastError());
}

void mps::kernel::sph::ComputeDFSPHConstantDensity(const mps::PhysicsParam& physParam, const mps::SPHParam& sph, const mps::SPHMaterialParam& sphMaterial, const mps::SpatialHashParam& sphHash, const mps::BoundaryParticleParam& boundaryParticle, const mps::MeshMaterialParam& boundaryParticleMaterial, const mps::SpatialHashParam& boundarhParticleHash)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	thrust::device_vector<REAL> d_error(1);
	thrust::host_vector<REAL> h_error(1);

	uint32_t l = 0u;
	while (l < 100u)
	{
		d_error.front() = static_cast<REAL>(0.0);

		ComputeDensityDelta_0(sph, sphMaterial, sphHash);
		ComputeDensityDelta_1(sph, sphMaterial, sphHash, boundaryParticle, boundaryParticleMaterial, boundarhParticleHash);
		ComputeCDStiffness_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize, nBlockSize * sizeof(REAL) >> >
			(physParam, sph, sphMaterial, thrust::raw_pointer_cast(d_error.data()));
		CUDA_CHECK(hipPeekAtLastError());

		h_error = d_error;
		h_error.front() /= static_cast<REAL>(sph.GetSize() + DBL_EPSILON);
		if (h_error.front() < 1.0e-4 && l >= 2u) break;

	#if DEBUG_PRINT
		std::stringstream ss;
		ss << "Pressure Error " << l << " : " << h_error.front() << std::endl;
		OutputDebugStringA(ss.str().c_str());
	#endif

		ApplyDFSPH_0(sph, sphMaterial, sphHash);
		ApplyDFSPH_1(sph, sphMaterial, sphHash, boundaryParticle, boundaryParticleMaterial, boundarhParticleHash);
		ApplyDFSPH_2(sph, sphMaterial);
		mps::kernel::UpdateVelocity(physParam, sph);
		l++;
	}
}

void mps::kernel::sph::ComputeDFSPHDivergenceFree(const mps::PhysicsParam& physParam, const mps::SPHParam& sph, const mps::SPHMaterialParam& sphMaterial, const mps::SpatialHashParam& sphHash, const mps::BoundaryParticleParam& boundaryParticle, const mps::MeshMaterialParam& boundaryParticleMaterial, const mps::SpatialHashParam& boundarhParticleHash)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	thrust::device_vector<REAL> d_error(1);
	thrust::host_vector<REAL> h_error(1);

	uint32_t l = 0u;
	while (l < 100u)
	{
		d_error.front() = static_cast<REAL>(0.0);

		ComputeDensityDelta_0(sph, sphMaterial, sphHash);
		ComputeDensityDelta_1(sph, sphMaterial, sphHash, boundaryParticle, boundaryParticleMaterial, boundarhParticleHash);
		ComputeDFStiffness_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize, nBlockSize * sizeof(REAL) >> >
			(physParam, sph, sphMaterial, thrust::raw_pointer_cast(d_error.data()));
		CUDA_CHECK(hipPeekAtLastError());

		h_error = d_error;
		h_error.front() /= static_cast<REAL>(sph.GetSize() + DBL_EPSILON);
		if (h_error.front() < 1.0e-3 && l >= 1u) break;

	#if DEBUG_PRINT
		std::stringstream ss;
		ss << "DivergenceFree Error " << l << " : " << h_error.front() << std::endl;
		OutputDebugStringA(ss.str().c_str());
	#endif

		ApplyDFSPH_0(sph, sphMaterial, sphHash);
		ApplyDFSPH_1(sph, sphMaterial, sphHash, boundaryParticle, boundaryParticleMaterial, boundarhParticleHash);
		ApplyDFSPH_2(sph, sphMaterial);
		mps::kernel::UpdateVelocity(physParam, sph);
		l++;
	}
}

void mps::kernel::sph::ApplyExplicitViscosity(const mps::SPHParam& sph, const mps::SPHMaterialParam& material, const mps::SpatialHashParam& hash)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	ApplyExplicitViscosity_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> >
		(sph, material, hash);
	CUDA_CHECK(hipPeekAtLastError());
}

void mps::kernel::sph::ApplyExplicitSurfaceTension(const mps::SPHParam& sph, const mps::SPHMaterialParam& material, const mps::SpatialHashParam& hash)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	ApplyExplicitSurfaceTension_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> >
		(sph, material, hash);
	CUDA_CHECK(hipPeekAtLastError());
}

void mps::kernel::sph::ApplyImplicitJacobiViscosity(const mps::PhysicsParam& physParam, const mps::SPHParam& sph, const mps::SPHMaterialParam& material, const mps::SpatialHashParam& hash)
{
	constexpr auto nBlockSize = 256u;
	constexpr auto nApplyBlockSize = 1024u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	thrust::device_vector<REAL> d_error(3);
	thrust::host_vector<REAL> h_error(3);
	thrust::device_vector<REAL> d_omega(3, 1.0);
	thrust::host_vector<REAL> h_omega(3, 1.0);

	std::vector<thrust::device_vector<REAL3>> d_tmp(3, thrust::device_vector<REAL3>{ sph.GetSize() });

	thrust::copy(thrust::device_pointer_cast(sph.GetVelocityArray()), thrust::device_pointer_cast(sph.GetVelocityArray() + sph.GetSize()), thrust::device_pointer_cast(sph.GetPredictVelArray()));
	thrust::copy(thrust::device_pointer_cast(sph.GetVelocityArray()), thrust::device_pointer_cast(sph.GetVelocityArray() + sph.GetSize()), thrust::device_pointer_cast(sph.GetPreviousVelArray()));

	REAL underRelax = 0.9;
	REAL omega = 1.0;
	REAL rho = 0.995;
	constexpr REAL delta = 0.005;

	uint32_t l = 0u;
	while (l < 100u)
	{
		uint32_t minErrorID = 0u;
		if (l < 10u)
		{
			d_error[0] = static_cast<REAL>(0.0);

			ComputeJacobiViscosity_0_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> > (
				physParam, sph, material, hash, thrust::raw_pointer_cast(d_tmp[0].data()));
			CUDA_CHECK(hipPeekAtLastError());

			ComputeJacobiError_0_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize, nBlockSize * sizeof(REAL) >> > (
				physParam, sph, material, hash, thrust::raw_pointer_cast(d_tmp[0].data()), thrust::raw_pointer_cast(d_error.data()));
			CUDA_CHECK(hipPeekAtLastError());

			h_error[0] = d_error[0];
		}
		else
		{
			if (l == 10u)
			{
				h_omega[0] = 2.0 / (2.0 - rho * rho);
				h_omega[1] = 2.0 / (2.0 - (rho + delta) * (rho + delta));
				h_omega[2] = 2.0 / (2.0 - (rho - delta) * (rho - delta));
				d_omega = h_omega;
			}
			else if (l > 10u)
			{
				h_omega[0] = 4.0 / (4.0 - rho * rho * omega);
				h_omega[1] = 4.0 / (4.0 - (rho + delta) * (rho + delta) * omega);
				h_omega[2] = 4.0 / (4.0 - (rho - delta) * (rho - delta) * omega);
				d_omega = h_omega;
			}

			h_error[0] = h_error[1] = h_error[2] = static_cast<REAL>(0.0);
			d_error = h_error;

			ComputeJacobiViscosity_1_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> > (
				physParam, sph, material, hash, thrust::raw_pointer_cast(d_tmp[0].data()), thrust::raw_pointer_cast(d_tmp[1].data()), thrust::raw_pointer_cast(d_tmp[2].data()),
				l, underRelax, thrust::raw_pointer_cast(d_omega.data()));
			CUDA_CHECK(hipPeekAtLastError());

			ComputeJacobiError_1_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize, nBlockSize * 3 * sizeof(REAL) >> > (
				physParam, sph, material, hash, thrust::raw_pointer_cast(d_tmp[0].data()), thrust::raw_pointer_cast(d_tmp[1].data()), thrust::raw_pointer_cast(d_tmp[2].data()),
				thrust::raw_pointer_cast(d_error.data()));
			CUDA_CHECK(hipPeekAtLastError());

			h_error = d_error;
			if (h_error[minErrorID] > h_error[1]) minErrorID = 1u;
			if (h_error[minErrorID] > h_error[2]) minErrorID = 2u;

			if (minErrorID == 1u) rho += delta;
			else if (minErrorID == 2u) rho -= delta;
			rho = std::min(std::max(0.9 + delta, rho), 1.0 - delta);
			omega = d_omega[minErrorID];
		}

		ApplyJacobiViscosity_kernel << < mcuda::util::DivUp(nSize, nApplyBlockSize), nApplyBlockSize >> >
			(sph, thrust::raw_pointer_cast(d_tmp[minErrorID].data()));
		CUDA_CHECK(hipPeekAtLastError());

		h_error[minErrorID] *= physParam.dt / static_cast<REAL>(nSize);
		if (h_error[minErrorID] < 1.0e-4) break;


	#if DEBUG_PRINT
		std::stringstream ss;
		ss << "Implicit Jacobi Viscosity Error " << l << " : " << h_error[minErrorID];
		ss << ", Omega/MinID " << l << " : " << omega << ", " << minErrorID << std::endl;
		OutputDebugStringA(ss.str().c_str());
	#endif
		l++;
	}

	thrust::copy(thrust::device_pointer_cast(sph.GetPredictVelArray()), thrust::device_pointer_cast(sph.GetPredictVelArray() + sph.GetSize()), thrust::device_pointer_cast(sph.GetVelocityArray()));
	std::stringstream ss;
	ss << "Implicit Jacobi Viscosity Loop : " << l << std::endl;
	OutputDebugStringA(ss.str().c_str());
}

//void mps::kernel::sph::ApplyImplicitJacobiViscosity(const mps::PhysicsParam& physParam, const mps::SPHParam& sph, const mps::SPHMaterialParam& material, const mps::SpatialHashParam& hash)
//{
//	constexpr auto nBlockSize = 256u;
//
//	const auto nSize = sph.GetSize();
//	if (nSize == 0) return;
//
//	thrust::device_vector<REAL> d_error(1);
//	thrust::host_vector<REAL> h_error(1);
//
//	thrust::device_vector<REAL3> d_tmp(sph.GetSize());
//
//	thrust::copy(thrust::device_pointer_cast(sph.GetVelocityArray()), thrust::device_pointer_cast(sph.GetVelocityArray() + sph.GetSize()), thrust::device_pointer_cast(sph.GetPredictVelArray()));
//	thrust::copy(thrust::device_pointer_cast(sph.GetVelocityArray()), thrust::device_pointer_cast(sph.GetVelocityArray() + sph.GetSize()), thrust::device_pointer_cast(sph.GetPreviousVelArray()));
//
//	constexpr REAL rho = 0.9962;
//	REAL underRelax = 1.0;
//	REAL omega = 1.0;
//
//	uint32_t l = 0u;
//	while (l < 300u)
//	{
//		if (l < 10u) { omega = 1.0; underRelax = 1.0; }
//		else if (l == 10u) { omega = 2.0 / (2.0 - rho * rho); underRelax = 0.9; }
//		else { omega = 4.0 / (4.0 - rho * rho * omega); underRelax = 0.8; }
//
//		d_error.front() = static_cast<REAL>(0.0);
//
//		ComputeJacobiViscosity_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize, nBlockSize * sizeof(REAL) >> >
//			(physParam, sph, material, hash, thrust::raw_pointer_cast(d_tmp.data()), omega, underRelax, thrust::raw_pointer_cast(d_error.data()));
//		CUDA_CHECK(hipPeekAtLastError());
//
//		h_error = d_error;
//		if (h_error.front() / static_cast<REAL>(nSize) < 1.0e-1 * physParam.dt) break;
//
//	#if DEBUG_PRINT
//		std::stringstream ss;
//		ss << "Implicit Jacobi Viscosity Error " << l << " : " << h_error.front() << std::endl;
//		ss << "Omega " << l << " : " << omega << std::endl;
//		OutputDebugStringA(ss.str().c_str());
//	#endif
//
//		ApplyJacobiViscosity_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> >
//			(sph, thrust::raw_pointer_cast(d_tmp.data()));
//		CUDA_CHECK(hipPeekAtLastError());
//		l++;
//	}
//
//	thrust::copy(thrust::device_pointer_cast(sph.GetPredictVelArray()), thrust::device_pointer_cast(sph.GetPredictVelArray() + sph.GetSize()), thrust::device_pointer_cast(sph.GetVelocityArray()));
//	std::stringstream ss;
//	ss << "Implicit Jacobi Viscosity Loop : " << l << std::endl;
//	OutputDebugStringA(ss.str().c_str());
//}

void mps::kernel::sph::ApplyImplicitGDViscosity(const mps::PhysicsParam& physParam, const mps::SPHParam& sph, const mps::SPHMaterialParam& material, const mps::SpatialHashParam& hash)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	thrust::device_vector<REAL3> d_R(sph.GetSize());
	thrust::host_vector<REAL3> h_R(sph.GetSize());
	thrust::device_vector<REAL> d_gama(2);
	thrust::host_vector<REAL> h_gama;

	thrust::copy(thrust::device_pointer_cast(sph.GetVelocityArray()), thrust::device_pointer_cast(sph.GetVelocityArray() + sph.GetSize()), thrust::device_pointer_cast(sph.GetPredictVelArray()));

	uint32_t l = 0u;
	while (l < 100u)
	{
		ComputeGDViscosityR_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> >
			(physParam, sph, material, hash, thrust::raw_pointer_cast(d_R.data()));
		CUDA_CHECK(hipPeekAtLastError());

		thrust::fill(d_gama.begin(), d_gama.end(), static_cast<REAL>(0.0));
		UpdateGDViscosityGama_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize, nBlockSize * 2 * sizeof(REAL) >> >
			(physParam, sph, material, hash, thrust::raw_pointer_cast(d_R.data()), thrust::raw_pointer_cast(d_gama.data()));
		CUDA_CHECK(hipPeekAtLastError());

		h_gama = d_gama;
		if (h_gama[0] < static_cast<REAL>(1.0e-2)) break;

	#if DEBUG_PRINT
		std::stringstream ss;
		ss << "Implicit GD Viscosity Error " << l << " : " << h_gama[0] << std::endl;
		OutputDebugStringA(ss.str().c_str());
	#endif

		const auto gama = h_gama[0] / h_gama[1];
		UpdateGDViscosity_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> >
			(sph, thrust::raw_pointer_cast(d_R.data()), gama);
		CUDA_CHECK(hipPeekAtLastError());
		l++;
	}

	thrust::copy(thrust::device_pointer_cast(sph.GetPredictVelArray()), thrust::device_pointer_cast(sph.GetPredictVelArray() + sph.GetSize()), thrust::device_pointer_cast(sph.GetVelocityArray()));
}

void mps::kernel::sph::ApplyImplicitCGViscosity(const mps::PhysicsParam& physParam, const mps::SPHParam& sph, const mps::SPHMaterialParam& material, const mps::SpatialHashParam& hash)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	thrust::device_vector<REAL3> d_R(sph.GetSize());
	thrust::device_vector<REAL3> d_V(sph.GetSize());
	thrust::device_vector<REAL3> d_Av(sph.GetSize());
	thrust::device_vector<REAL> d_param(2);
	thrust::host_vector<REAL> h_param;

	thrust::copy(thrust::device_pointer_cast(sph.GetVelocityArray()), thrust::device_pointer_cast(sph.GetVelocityArray() + sph.GetSize()), thrust::device_pointer_cast(sph.GetPredictVelArray()));
	ComputeGDViscosityR_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> >
		(physParam, sph, material, hash, thrust::raw_pointer_cast(d_R.data()));
	CUDA_CHECK(hipPeekAtLastError());
	thrust::copy(d_R.begin(), d_R.end(), d_V.begin());

	constexpr REAL factor = 0.1;
	uint32_t l = 0u;
	while (l < 100u)
	{
		ComputeCGViscosityAv_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> >
			(physParam, sph, material, hash, thrust::raw_pointer_cast(d_V.data()), thrust::raw_pointer_cast(d_Av.data()), factor);
		CUDA_CHECK(hipPeekAtLastError());

		thrust::fill(d_param.begin(), d_param.end(), static_cast<REAL>(0.0));
		UpdateCGViscosityAlphaParam_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize, nBlockSize * 2 * sizeof(REAL) >> >
			(sph, thrust::raw_pointer_cast(d_R.data()), thrust::raw_pointer_cast(d_V.data()), thrust::raw_pointer_cast(d_Av.data()), thrust::raw_pointer_cast(d_param.data()));
		CUDA_CHECK(hipPeekAtLastError());

		h_param = d_param;
		if (h_param[0] < static_cast<REAL>(1.0e-2)) break;

	#if DEBUG_PRINT
		std::stringstream ss;
		ss << "Implicit GD Viscosity Error " << l << " : " << h_param[0] << std::endl;
		OutputDebugStringA(ss.str().c_str());
	#endif

		const auto alpha = h_param[0] / h_param[1];
		UpdateCGViscosityXR_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> >
			(sph, thrust::raw_pointer_cast(d_R.data()), thrust::raw_pointer_cast(d_V.data()), thrust::raw_pointer_cast(d_Av.data()), alpha);
		CUDA_CHECK(hipPeekAtLastError());

		d_param[0] = static_cast<REAL>(0.0);
		UpdateCGViscosityBetaParam_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize, nBlockSize * sizeof(REAL) >> >
			(sph, thrust::raw_pointer_cast(d_R.data()), thrust::raw_pointer_cast(d_param.data()));
		CUDA_CHECK(hipPeekAtLastError());

		h_param[1] = d_param[0];
		const auto beta = h_param[1] / h_param[0];
		UpdateCGViscosityV_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> >
			(sph, thrust::raw_pointer_cast(d_R.data()), thrust::raw_pointer_cast(d_V.data()), beta);
		CUDA_CHECK(hipPeekAtLastError());
		l++;
	}

	thrust::copy(thrust::device_pointer_cast(sph.GetPredictVelArray()), thrust::device_pointer_cast(sph.GetPredictVelArray() + sph.GetSize()), thrust::device_pointer_cast(sph.GetVelocityArray()));
	std::stringstream ss;
	ss << "Implicit GD Viscosity Loop : " << l << std::endl;
	OutputDebugStringA(ss.str().c_str());
}

void mps::kernel::sph::DensityColorTest(const mps::SPHParam& sph, const mps::SPHMaterialParam& material)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	DensityColorTest_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> >
		(sph, material);
	CUDA_CHECK(hipPeekAtLastError());
}