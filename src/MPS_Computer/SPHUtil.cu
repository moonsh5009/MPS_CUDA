#include "hip/hip_runtime.h"
#include "stdafx.h"
#include "SPHUtil.cuh"
#include "AdvectUtil.h"

#include <thrust/host_vector.h>
#include <thrust/extrema.h>

#define DEBUG_PRINT		0

void mps::kernel::sph::ComputeBoundaryParticleVolume_0(const mps::BoundaryParticleParam& boundaryParticle, const mps::MeshMaterial* pMaterial, const mps::SpatialHash* pHash)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = boundaryParticle.GetSize();
	if (nSize == 0) return;
	
	const auto optNei = pHash->GetNeighborhood(boundaryParticle);
	if (!optNei) return;
	const auto& [pNei, pNeiIdx] = optNei.value();

	ComputeBoundaryParticleVolume_0_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> > (
		boundaryParticle, pMaterial->GetParam(), pNei, pNeiIdx);
	CUDA_CHECK(hipPeekAtLastError());
}
void mps::kernel::sph::ComputeBoundaryParticleVolume_1(
	const mps::BoundaryParticleParam& boundaryParticle, const mps::MeshMaterial* pMaterial,
	const mps::BoundaryParticleParam& refBoundaryParticle, const mps::MeshMaterial* pRefMaterial,
	const mps::SpatialHash* pRefHash)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = boundaryParticle.GetSize();
	if (nSize == 0) return;

	const auto optNei = pRefHash->GetNeighborhood(refBoundaryParticle);
	if (!optNei) return;
	const auto& [pNei, pNeiIdx] = optNei.value();

	ComputeBoundaryParticleVolume_1_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> > (
		boundaryParticle, pMaterial->GetParam(), refBoundaryParticle, pRefMaterial->GetParam(), pNei, pNeiIdx);
	CUDA_CHECK(hipPeekAtLastError());
}
void mps::kernel::sph::ComputeBoundaryParticleVolume_2(const mps::BoundaryParticleParam& boundaryParticle, const mps::MeshMaterial* pMaterial)
{
	constexpr auto nBlockSize = 1024u;

	const auto nSize = boundaryParticle.GetSize();
	if (nSize == 0) return;

	ComputeBoundaryParticleVolume_2_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> > (
		boundaryParticle, pMaterial->GetParam());
	CUDA_CHECK(hipPeekAtLastError());
}

void mps::kernel::sph::ComputeDensity_0(const mps::SPHParam& sph, const mps::SPHMaterial* pMaterial, const mps::SpatialHash* pHash)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	const auto optNei = pHash->GetNeighborhood(sph);
	if (!optNei) return;
	const auto& [pNei, pNeiIdx] = optNei.value();

	ComputeDensity_0_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> > (
		sph, pMaterial->GetParam(), pNei, pNeiIdx);
	CUDA_CHECK(hipPeekAtLastError());
}
void mps::kernel::sph::ComputeDensity_1(
	const mps::SPHParam& sph, const mps::SPHMaterial* pSPHMaterial,
	const mps::BoundaryParticleParam& boundaryParticle, const mps::MeshMaterial* pBoundaryParticleMaterial,
	const mps::SpatialHash* pSPHHash)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	const auto optNei = pSPHHash->GetNeighborhood(boundaryParticle);
	if (!optNei) return;
	const auto& [pNei, pNeiIdx] = optNei.value();

	ComputeDensity_1_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> > (
		sph, pSPHMaterial->GetParam(), boundaryParticle, pBoundaryParticleMaterial->GetParam(), pNei, pNeiIdx);
	CUDA_CHECK(hipPeekAtLastError());
}
void mps::kernel::sph::ComputeDensity_2(const mps::SPHParam& sph, const mps::SPHMaterial* pMaterial)
{
	constexpr auto nBlockSize = 1024u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	ComputeDensity_2_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> > (
		sph, pMaterial->GetParam());
	CUDA_CHECK(hipPeekAtLastError());
}

void mps::kernel::sph::ComputeDFSPHFactor_0(const mps::SPHParam& sph, const mps::SPHMaterial* pMaterial, const mps::SpatialHash* pHash)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	const auto optNei = pHash->GetNeighborhood(sph);
	if (!optNei) return;
	const auto& [pNei, pNeiIdx] = optNei.value();

	ComputeDFSPHFactor_0_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> > (
		sph, pMaterial->GetParam(), pNei, pNeiIdx);
	CUDA_CHECK(hipPeekAtLastError());
}
void mps::kernel::sph::ComputeDFSPHFactor_1(
	const mps::SPHParam& sph, const mps::SPHMaterial* pSPHMaterial,
	const mps::BoundaryParticleParam& boundaryParticle, const mps::MeshMaterial* pBoundaryParticleMaterial,
	const mps::SpatialHash* pSPHHash)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	const auto optNei = pSPHHash->GetNeighborhood(boundaryParticle);
	if (!optNei) return;
	const auto& [pNei, pNeiIdx] = optNei.value();

	ComputeDFSPHFactor_1_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> > (
		sph, pSPHMaterial->GetParam(), boundaryParticle, pBoundaryParticleMaterial->GetParam(), pNei, pNeiIdx);
	CUDA_CHECK(hipPeekAtLastError());
}
void mps::kernel::sph::ComputeDFSPHFactor_2(const mps::SPHParam& sph, const mps::SPHMaterial* pMaterial)
{
	constexpr auto nBlockSize = 1024u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	ComputeDFSPHFactor_2_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> >
		(sph, pMaterial->GetParam());
	CUDA_CHECK(hipPeekAtLastError());
}

void mps::kernel::sph::ComputeDensityDelta_0(const mps::SPHParam& sph, const mps::SPHMaterial* pMaterial, const mps::SpatialHash* pHash)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	const auto optNei = pHash->GetNeighborhood(sph);
	if (!optNei) return;
	const auto& [pNei, pNeiIdx] = optNei.value();

	ComputeDensityDelta_0_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> > (
		sph, pMaterial->GetParam(), pNei, pNeiIdx);
	CUDA_CHECK(hipPeekAtLastError());
}
void mps::kernel::sph::ComputeDensityDelta_1(
	const mps::SPHParam& sph, const mps::SPHMaterial* pSPHMaterial,
	const mps::BoundaryParticleParam& boundaryParticle, const mps::MeshMaterial* pBoundaryParticleMaterial,
	const mps::SpatialHash* pSPHHash)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	const auto optNei = pSPHHash->GetNeighborhood(boundaryParticle);
	if (!optNei) return;
	const auto& [pNei, pNeiIdx] = optNei.value();

	ComputeDensityDelta_1_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> > (
		sph, pSPHMaterial->GetParam(), boundaryParticle, pBoundaryParticleMaterial->GetParam(), pNei, pNeiIdx);
	CUDA_CHECK(hipPeekAtLastError());
}

void mps::kernel::sph::ApplyDFSPH_0(const mps::SPHParam& sph, const mps::SPHMaterial* pMaterial, const mps::SpatialHash* pHash)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	const auto optNei = pHash->GetNeighborhood(sph);
	if (!optNei) return;
	const auto& [pNei, pNeiIdx] = optNei.value();

	ApplyDFSPH_0_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> > (
		sph, pMaterial->GetParam(), pNei, pNeiIdx);
	CUDA_CHECK(hipPeekAtLastError());
}
void mps::kernel::sph::ApplyDFSPH_1(
	const mps::SPHParam& sph, const mps::SPHMaterial* pSPHMaterial,
	const mps::BoundaryParticleParam& boundaryParticle, const mps::MeshMaterial* pBoundaryParticleMaterial,
	const mps::SpatialHash* pSPHHash)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	const auto optNei = pSPHHash->GetNeighborhood(boundaryParticle);
	if (!optNei) return;
	const auto& [pNei, pNeiIdx] = optNei.value();

	ApplyDFSPH_1_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> > (
		sph, pSPHMaterial->GetParam(), boundaryParticle, pBoundaryParticleMaterial->GetParam(), pNei, pNeiIdx);
	CUDA_CHECK(hipPeekAtLastError());
}
void mps::kernel::sph::ApplyDFSPH_2(const mps::SPHParam& sph, const mps::SPHMaterial* pMaterial)
{
	constexpr auto nBlockSize = 1024u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	ApplyDFSPH_2_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> > (
		sph, pMaterial->GetParam());
	CUDA_CHECK(hipPeekAtLastError());
}

void mps::kernel::sph::ComputeDFSPHConstantDensity(const mps::PhysicsParam& physParam,
	const mps::SPHParam& sph, const mps::SPHMaterial* pSPHMaterial,
	const mps::BoundaryParticleParam& boundaryParticle, const mps::MeshMaterial* pBoundaryParticleMaterial,
	const mps::SpatialHash* pSPHHash)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	thrust::device_vector<REAL> d_error(1);
	thrust::host_vector<REAL> h_error(1);

	uint32_t l = 0u;
	while (l < 100u)
	{
		d_error.front() = static_cast<REAL>(0.0);

		ComputeDensityDelta_0(sph, pSPHMaterial, pSPHHash);
		ComputeDensityDelta_1(sph, pSPHMaterial, boundaryParticle, pBoundaryParticleMaterial, pSPHHash);
		ComputeCDStiffness_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize, nBlockSize * sizeof(REAL) >> > (
			physParam, sph, pSPHMaterial->GetParam(), thrust::raw_pointer_cast(d_error.data()));
		CUDA_CHECK(hipPeekAtLastError());

		h_error = d_error;
		h_error.front() /= static_cast<REAL>(sph.GetSize() + DBL_EPSILON);
		if (h_error.front() < 1.0e-4 && l >= 2u) break;

	#if DEBUG_PRINT
		std::stringstream ss;
		ss << "Pressure Error " << l << " : " << h_error.front() << std::endl;
		OutputDebugStringA(ss.str().c_str());
	#endif

		ApplyDFSPH_0(sph, pSPHMaterial, pSPHHash);
		ApplyDFSPH_1(sph, pSPHMaterial, boundaryParticle, pBoundaryParticleMaterial, pSPHHash);
		ApplyDFSPH_2(sph, pSPHMaterial);
		mps::kernel::UpdateVelocity(physParam, sph);
		l++;
	}
}
void mps::kernel::sph::ComputeDFSPHDivergenceFree(const mps::PhysicsParam& physParam,
	const mps::SPHParam& sph, const mps::SPHMaterial* pSPHMaterial,
	const mps::BoundaryParticleParam& boundaryParticle, const mps::MeshMaterial* pBoundaryParticleMaterial,
	const mps::SpatialHash* pSPHHash)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	thrust::device_vector<REAL> d_error(1);
	thrust::host_vector<REAL> h_error(1);

	uint32_t l = 0u;
	while (l < 100u)
	{
		d_error.front() = static_cast<REAL>(0.0);

		ComputeDensityDelta_0(sph, pSPHMaterial, pSPHHash);
		ComputeDensityDelta_1(sph, pSPHMaterial, boundaryParticle, pBoundaryParticleMaterial, pSPHHash);
		ComputeDFStiffness_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize, nBlockSize * sizeof(REAL) >> > (
			physParam, sph, pSPHMaterial->GetParam(), thrust::raw_pointer_cast(d_error.data()));
		CUDA_CHECK(hipPeekAtLastError());

		h_error = d_error;
		h_error.front() /= static_cast<REAL>(sph.GetSize() + DBL_EPSILON);
		if (h_error.front() < 1.0e-3 && l >= 1u) break;

	#if DEBUG_PRINT
		std::stringstream ss;
		ss << "DivergenceFree Error " << l << " : " << h_error.front() << std::endl;
		OutputDebugStringA(ss.str().c_str());
	#endif

		ApplyDFSPH_0(sph, pSPHMaterial, pSPHHash);
		ApplyDFSPH_1(sph, pSPHMaterial, boundaryParticle, pBoundaryParticleMaterial, pSPHHash);
		ApplyDFSPH_2(sph, pSPHMaterial);
		mps::kernel::UpdateVelocity(physParam, sph);
		l++;
	}
}

void mps::kernel::sph::ApplyExplicitViscosity(const mps::SPHParam& sph, const mps::SPHMaterial* pMaterial, const mps::SpatialHash* pHash)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	const auto optNei = pHash->GetNeighborhood(sph);
	if (!optNei) return;
	const auto& [pNei, pNeiIdx] = optNei.value();

	ApplyExplicitViscosity_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> > (
		sph, pMaterial->GetParam(), pNei, pNeiIdx);
	CUDA_CHECK(hipPeekAtLastError());
}

void mps::kernel::sph::ApplyExplicitSurfaceTension(const mps::SPHParam& sph, const mps::SPHMaterial* pMaterial, const mps::SpatialHash* pHash)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	const auto optNei = pHash->GetNeighborhood(sph);
	if (!optNei) return;
	const auto& [pNei, pNeiIdx] = optNei.value();

	ApplyExplicitSurfaceTension_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> > (
		sph, pMaterial->GetParam(), pNei, pNeiIdx);
	CUDA_CHECK(hipPeekAtLastError());
}

void mps::kernel::sph::ApplyImplicitJacobiViscosity(const mps::PhysicsParam& physParam,
	const mps::SPHParam& sph, const mps::SPHMaterial* pMaterial, const mps::SpatialHash* pHash)
{
	constexpr auto nBlockSize = 256u;
	constexpr auto nApplyBlockSize = 1024u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	const auto optNei = pHash->GetNeighborhood(sph);
	if (!optNei) return;
	const auto& [pNei, pNeiIdx] = optNei.value();

	thrust::device_vector<REAL> d_error(3);
	thrust::host_vector<REAL> h_error(3);
	thrust::device_vector<REAL> d_omega(3, 1.0);
	thrust::host_vector<REAL> h_omega(3, 1.0);

	std::vector<thrust::device_vector<REAL3>> d_tmp(3, thrust::device_vector<REAL3>{ sph.GetSize() });

	thrust::copy(thrust::device_pointer_cast(sph.GetVelocityArray()), thrust::device_pointer_cast(sph.GetVelocityArray() + sph.GetSize()), thrust::device_pointer_cast(sph.GetPredictVelArray()));
	thrust::copy(thrust::device_pointer_cast(sph.GetVelocityArray()), thrust::device_pointer_cast(sph.GetVelocityArray() + sph.GetSize()), thrust::device_pointer_cast(sph.GetPreviousVelArray()));

	REAL underRelax = 0.9;
	REAL omega = 1.0;
	REAL rho = 0.995;
	constexpr REAL delta = 0.005;

	uint32_t l = 0u;
	while (l < 100u)
	{
		uint32_t minErrorID = 0u;
		if (l < 10u)
		{
			d_error[0] = static_cast<REAL>(0.0);

			ComputeJacobiViscosity_0_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> > (
				physParam, sph, pMaterial->GetParam(), pNei, pNeiIdx, thrust::raw_pointer_cast(d_tmp[0].data()));
			CUDA_CHECK(hipPeekAtLastError());

			ComputeJacobiError_0_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize, nBlockSize * sizeof(REAL) >> > (
				physParam, sph, pMaterial->GetParam(), pNei, pNeiIdx, thrust::raw_pointer_cast(d_tmp[0].data()), thrust::raw_pointer_cast(d_error.data()));
			CUDA_CHECK(hipPeekAtLastError());

			h_error[0] = d_error[0];
		}
		else
		{
			if (l == 10u)
			{
				h_omega[0] = 2.0 / (2.0 - rho * rho);
				h_omega[1] = 2.0 / (2.0 - (rho + delta) * (rho + delta));
				h_omega[2] = 2.0 / (2.0 - (rho - delta) * (rho - delta));
				d_omega = h_omega;
			}
			else if (l > 10u)
			{
				h_omega[0] = 4.0 / (4.0 - rho * rho * omega);
				h_omega[1] = 4.0 / (4.0 - (rho + delta) * (rho + delta) * omega);
				h_omega[2] = 4.0 / (4.0 - (rho - delta) * (rho - delta) * omega);
				d_omega = h_omega;
			}

			h_error[0] = h_error[1] = h_error[2] = static_cast<REAL>(0.0);
			d_error = h_error;

			ComputeJacobiViscosity_1_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> > (
				physParam, sph, pMaterial->GetParam(), pNei, pNeiIdx,
				thrust::raw_pointer_cast(d_tmp[0].data()), thrust::raw_pointer_cast(d_tmp[1].data()), thrust::raw_pointer_cast(d_tmp[2].data()),
				l, underRelax, thrust::raw_pointer_cast(d_omega.data()));
			CUDA_CHECK(hipPeekAtLastError());

			ComputeJacobiError_1_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize, nBlockSize * 3 * sizeof(REAL) >> > (
				physParam, sph, pMaterial->GetParam(), pNei, pNeiIdx,
				thrust::raw_pointer_cast(d_tmp[0].data()), thrust::raw_pointer_cast(d_tmp[1].data()), thrust::raw_pointer_cast(d_tmp[2].data()),
				thrust::raw_pointer_cast(d_error.data()));
			CUDA_CHECK(hipPeekAtLastError());

			h_error = d_error;
			if (h_error[minErrorID] > h_error[1]) minErrorID = 1u;
			if (h_error[minErrorID] > h_error[2]) minErrorID = 2u;

			if (minErrorID == 1u) rho += delta;
			else if (minErrorID == 2u) rho -= delta;
			rho = std::min(std::max(0.5 + delta, rho), 1.0 - delta);
			omega = d_omega[minErrorID];
		}

		ApplyJacobiViscosity_kernel << < mcuda::util::DivUp(nSize, nApplyBlockSize), nApplyBlockSize >> > (
			sph, thrust::raw_pointer_cast(d_tmp[minErrorID].data()));
		CUDA_CHECK(hipPeekAtLastError());

		h_error[minErrorID] *= physParam.dt / static_cast<REAL>(nSize);
		if (h_error[minErrorID] < 1.0e-4) break;


	#if DEBUG_PRINT
		std::stringstream ss;
		ss << "Implicit Jacobi Viscosity Error " << l << " : " << h_error[minErrorID];
		ss << ", Omega/MinID " << l << " : " << omega << ", " << minErrorID << std::endl;
		OutputDebugStringA(ss.str().c_str());
	#endif
		l++;
	}

	thrust::copy(thrust::device_pointer_cast(sph.GetPredictVelArray()), thrust::device_pointer_cast(sph.GetPredictVelArray() + sph.GetSize()), thrust::device_pointer_cast(sph.GetVelocityArray()));
	std::stringstream ss;
	ss << "Implicit Jacobi Viscosity Loop : " << l << std::endl;
	OutputDebugStringA(ss.str().c_str());
}

void mps::kernel::sph::ApplyImplicitGDViscosity(const mps::PhysicsParam& physParam,
	const mps::SPHParam& sph, const mps::SPHMaterial* pMaterial, const mps::SpatialHash* pHash)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	const auto optNei = pHash->GetNeighborhood(sph);
	if (!optNei) return;
	const auto& [pNei, pNeiIdx] = optNei.value();

	thrust::device_vector<REAL3> d_R(sph.GetSize());
	thrust::host_vector<REAL3> h_R(sph.GetSize());
	thrust::device_vector<REAL> d_gama(2);
	thrust::host_vector<REAL> h_gama;

	thrust::copy(thrust::device_pointer_cast(sph.GetVelocityArray()), thrust::device_pointer_cast(sph.GetVelocityArray() + sph.GetSize()), thrust::device_pointer_cast(sph.GetPredictVelArray()));

	uint32_t l = 0u;
	while (l < 100u)
	{
		ComputeGDViscosityR_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> > (
			physParam, sph, pMaterial->GetParam(), pNei, pNeiIdx, thrust::raw_pointer_cast(d_R.data()));
		CUDA_CHECK(hipPeekAtLastError());

		thrust::fill(d_gama.begin(), d_gama.end(), static_cast<REAL>(0.0));
		UpdateGDViscosityGama_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize, nBlockSize * 2 * sizeof(REAL) >> > (
			physParam, sph, pMaterial->GetParam(), pNei, pNeiIdx, thrust::raw_pointer_cast(d_R.data()), thrust::raw_pointer_cast(d_gama.data()));
		CUDA_CHECK(hipPeekAtLastError());

		h_gama = d_gama;
		if (h_gama[0] < static_cast<REAL>(1.0e-2)) break;

	#if DEBUG_PRINT
		std::stringstream ss;
		ss << "Implicit GD Viscosity Error " << l << " : " << h_gama[0] << std::endl;
		OutputDebugStringA(ss.str().c_str());
	#endif

		const auto gama = h_gama[0] / h_gama[1];
		UpdateGDViscosity_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> > (
			sph, thrust::raw_pointer_cast(d_R.data()), gama);
		CUDA_CHECK(hipPeekAtLastError());
		l++;
	}

	thrust::copy(thrust::device_pointer_cast(sph.GetPredictVelArray()), thrust::device_pointer_cast(sph.GetPredictVelArray() + sph.GetSize()), thrust::device_pointer_cast(sph.GetVelocityArray()));
}

void mps::kernel::sph::ApplyImplicitCGViscosity(const mps::PhysicsParam& physParam,
	const mps::SPHParam& sph, const mps::SPHMaterial* pMaterial, const mps::SpatialHash* pHash)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	const auto optNei = pHash->GetNeighborhood(sph);
	if (!optNei) return;
	const auto& [pNei, pNeiIdx] = optNei.value();

	thrust::device_vector<REAL3> d_R(sph.GetSize());
	thrust::device_vector<REAL3> d_V(sph.GetSize());
	thrust::device_vector<REAL3> d_Av(sph.GetSize());
	thrust::device_vector<REAL> d_param(2);
	thrust::host_vector<REAL> h_param;

	thrust::copy(thrust::device_pointer_cast(sph.GetVelocityArray()), thrust::device_pointer_cast(sph.GetVelocityArray() + sph.GetSize()), thrust::device_pointer_cast(sph.GetPredictVelArray()));
	ComputeGDViscosityR_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> > (
		physParam, sph, pMaterial->GetParam(), pNei, pNeiIdx, thrust::raw_pointer_cast(d_R.data()));
	CUDA_CHECK(hipPeekAtLastError());
	thrust::copy(d_R.begin(), d_R.end(), d_V.begin());

	REAL factor = 0.01;
	uint32_t l = 0u;
	while (l < 100u)
	{
		if (l < 18u)		factor = 0.01;
		else if (l == 18u)	factor = 0.1;
		else				factor += l * 0.016;

		ComputeCGViscosityAv_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> > (
			physParam, sph, pMaterial->GetParam(), pNei, pNeiIdx, thrust::raw_pointer_cast(d_V.data()), thrust::raw_pointer_cast(d_Av.data()), factor);
		CUDA_CHECK(hipPeekAtLastError());

		thrust::fill(d_param.begin(), d_param.end(), static_cast<REAL>(0.0));
		UpdateCGViscosityAlphaParam_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize, nBlockSize * 2 * sizeof(REAL) >> > (
			sph, thrust::raw_pointer_cast(d_R.data()), thrust::raw_pointer_cast(d_V.data()), thrust::raw_pointer_cast(d_Av.data()), thrust::raw_pointer_cast(d_param.data()));
		CUDA_CHECK(hipPeekAtLastError());

		h_param = d_param;
		if (h_param[0] < static_cast<REAL>(1.0e-2)) break;

	#if DEBUG_PRINT
		std::stringstream ss;
		ss << "Implicit GD Viscosity Error " << l << " : " << h_param[0] << std::endl;
		OutputDebugStringA(ss.str().c_str());
	#endif

		const auto alpha = h_param[0] / h_param[1];
		UpdateCGViscosityXR_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> > (
			sph, thrust::raw_pointer_cast(d_R.data()), thrust::raw_pointer_cast(d_V.data()), thrust::raw_pointer_cast(d_Av.data()), alpha);
		CUDA_CHECK(hipPeekAtLastError());

		d_param[0] = static_cast<REAL>(0.0);
		UpdateCGViscosityBetaParam_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize, nBlockSize * sizeof(REAL) >> > (
			sph, thrust::raw_pointer_cast(d_R.data()), thrust::raw_pointer_cast(d_param.data()));
		CUDA_CHECK(hipPeekAtLastError());

		h_param[1] = d_param[0];
		const auto beta = h_param[1] / h_param[0];
		UpdateCGViscosityV_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> > (
			sph, thrust::raw_pointer_cast(d_R.data()), thrust::raw_pointer_cast(d_V.data()), beta);
		CUDA_CHECK(hipPeekAtLastError());
		l++;
	}

	thrust::copy(thrust::device_pointer_cast(sph.GetPredictVelArray()), thrust::device_pointer_cast(sph.GetPredictVelArray() + sph.GetSize()), thrust::device_pointer_cast(sph.GetVelocityArray()));
	std::stringstream ss;
	ss << "Implicit GD Viscosity Loop : " << l << std::endl;
	OutputDebugStringA(ss.str().c_str());
}

void mps::kernel::sph::DensityColorTest(const mps::SPHParam& sph, const mps::SPHMaterial* pMaterial)
{
	constexpr auto nBlockSize = 256u;

	const auto nSize = sph.GetSize();
	if (nSize == 0) return;

	DensityColorTest_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> > (
		sph, pMaterial->GetParam());
	CUDA_CHECK(hipPeekAtLastError());
}