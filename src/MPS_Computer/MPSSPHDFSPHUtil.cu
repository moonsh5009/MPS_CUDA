#include "stdafx.h"
#include "MPSSPHDFSPHUtil.cuh"
#include <thrust/extrema.h>
#include <thrust/host_vector.h>

#include "../MPS_Object/MPSSPHParam.h"
#include "../MPS_Object/MPSBoundaryParticleParam.h"
#include "../MPS_Object/MPSSpatialHash.h"

#include "MPSAdvectUtil.h"

void mps::kernel::SPH::ComputeDFSPHFactorSub(
	const mps::SPHMaterialParam& sphMaterial,
	const mps::SPHParam& sph,
	const mps::NeiParam& nei)
{
	if (sph.size == 0) return;
	
	ComputeDFSPHFactor_kernel << < mcuda::util::DivUp(sph.size, nBlockSize), nBlockSize >> > (
		sphMaterial,
		sph.pPosition,
		sph.pRadius,
		sph.pTempVec3,
		sph.pFactorDFSPH,
		sph.size,
		nei.pID,
		nei.pIdx);
	CUDA_CHECK(hipPeekAtLastError());
}

void mps::kernel::SPH::ComputeDFSPHFactorSub(
	const mps::SPHParam& sph,
	const mps::NeiParam& nei,
	const mps::SPHMaterialParam& refSPHMaterial,
	const mps::SPHParam& refSPH)
{
	if (sph.size == 0) return;

	ComputeDFSPHFactor_kernel << < mcuda::util::DivUp(sph.size, nBlockSize), nBlockSize >> > (
		sph.pPosition,
		sph.pRadius,
		sph.pTempVec3,
		sph.pFactorDFSPH,
		sph.size,
		refSPHMaterial,
		refSPH.pPosition,
		refSPH.pRadius,
		nei.pID,
		nei.pIdx);
	CUDA_CHECK(hipPeekAtLastError());
}

void mps::kernel::SPH::ComputeDFSPHFactorSub(
	const mps::SPHParam& sph,
	const mps::NeiParam& nei,
	const mps::BoundaryParticleParam& boundaryParticle)
{
	if (sph.size == 0) return;

	ComputeDFSPHFactor_kernel << < mcuda::util::DivUp(sph.size, nBlockSize), nBlockSize >> > (
		sph.pPosition,
		sph.pRadius,
		sph.pTempVec3,
		sph.size,
		boundaryParticle.pPosition,
		boundaryParticle.pRadius,
		boundaryParticle.pVolume,
		nei.pID,
		nei.pIdx);
	CUDA_CHECK(hipPeekAtLastError());
}

void mps::kernel::SPH::ComputeDFSPHFactorFinal(
	const mps::SPHParam& sph)
{
	if (sph.size == 0) return;

	ComputeDFSPHFactorFinal_kernel << < mcuda::util::DivUp(sph.size, nFullBlockSize), nFullBlockSize >> > (
		sph.pTempVec3,
		sph.pFactorDFSPH,
		sph.size);
	CUDA_CHECK(hipPeekAtLastError());
}

void mps::kernel::SPH::ComputeDensityDeltaSub(
	const mps::SPHMaterialParam& sphMaterial,
	const mps::SPHParam& sph,
	const mps::NeiParam& nei,
	long long stream)
{
	if (sph.size == 0) return;

	ComputeDensityDelta_kernel << < mcuda::util::DivUp(sph.size, nBlockSize), nBlockSize, 0, reinterpret_cast<hipStream_t>(stream) >> > (
		sphMaterial,
		sph.pPosition,
		sph.pVelocity,
		sph.pRadius,
		sph.pTempReal,
		sph.size,
		nei.pID,
		nei.pIdx);
	CUDA_CHECK(hipPeekAtLastError());
}

void mps::kernel::SPH::ComputeDensityDeltaSub(
	const mps::SPHParam& sph,
	const mps::NeiParam& nei,
	const mps::SPHMaterialParam& refSPHMaterial,
	const mps::SPHParam& refSPH,
	long long stream)
{
	if (sph.size == 0) return;

	ComputeDensityDelta_kernel << < mcuda::util::DivUp(sph.size, nBlockSize), nBlockSize, 0, reinterpret_cast<hipStream_t>(stream) >> > (
		sph.pPosition,
		sph.pVelocity,
		sph.pRadius,
		sph.pTempReal,
		sph.size,
		refSPHMaterial,
		refSPH.pPosition,
		refSPH.pVelocity,
		refSPH.pRadius,
		nei.pID,
		nei.pIdx);
	CUDA_CHECK(hipPeekAtLastError());
}

void mps::kernel::SPH::ComputeDensityDeltaSub(
	const mps::SPHParam& sph,
	const mps::NeiParam& nei,
	const mps::BoundaryParticleParam& boundaryParticle,
	long long stream)
{
	if (sph.size == 0) return;

	ComputeDensityDelta_kernel << < mcuda::util::DivUp(sph.size, nBlockSize), nBlockSize, 0, reinterpret_cast<hipStream_t>(stream) >> > (
		sph.pPosition,
		sph.pVelocity,
		sph.pRadius,
		sph.pTempReal,
		sph.size,
		boundaryParticle.pPosition,
		boundaryParticle.pVelocity,
		boundaryParticle.pRadius,
		boundaryParticle.pVolume,
		nei.pID,
		nei.pIdx);
	CUDA_CHECK(hipPeekAtLastError());
}

void mps::kernel::SPH::ComputeDFSPHConstantDensitySub(
	const mps::PhysicsParam& physParam,
	const mps::SPHMaterialParam& sphMaterial,
	const mps::SPHParam& sph,
	REAL* sumError)
{
	if (sph.size == 0) return;

	ComputeCDStiffness_kernel << < mcuda::util::DivUp(sph.size, nBlockSize), nBlockSize, nBlockSize * sizeof(REAL) >> > (
		physParam,
		sphMaterial,
		sph.pDensity,
		sph.pFactorDFSPH,
		sph.pTempReal,
		sph.pPressure,
		sph.size,
		sumError);
	CUDA_CHECK(hipPeekAtLastError());
}

void mps::kernel::SPH::ComputeDFSPHDivergenceFreeSub(
	const mps::PhysicsParam& physParam,
	const mps::SPHMaterialParam& sphMaterial,
	const mps::SPHParam& sph,
	REAL* sumError)
{
	if (sph.size == 0) return;

	ComputeDFStiffness_kernel << < mcuda::util::DivUp(sph.size, nBlockSize), nBlockSize, nBlockSize * sizeof(REAL) >> > (
		physParam,
		sphMaterial,
		sph.pDensity,
		sph.pFactorDFSPH,
		sph.pTempReal,
		sph.pPressure,
		sph.size,
		sumError);
	CUDA_CHECK(hipPeekAtLastError());
}

void mps::kernel::SPH::ApplyDFSPHSub(
	const mps::SPHMaterialParam& sphMaterial,
	const mps::SPHParam& sph,
	const mps::NeiParam& nei,
	long long stream)
{
	if (sph.size == 0) return;

	ApplyDFSPH_kernel << < mcuda::util::DivUp(sph.size, nBlockSize), nBlockSize, 0, reinterpret_cast<hipStream_t>(stream) >> > (
		sphMaterial,
		sph.pPosition,
		sph.pRadius,
		sph.pPressure,
		sph.pForce,
		sph.size,
		nei.pID,
		nei.pIdx);
	CUDA_CHECK(hipPeekAtLastError());
}

void mps::kernel::SPH::ApplyDFSPHSub(
	const mps::SPHParam& sph,
	const mps::NeiParam& nei,
	const mps::SPHMaterialParam& pRefSPHMaterial,
	const mps::SPHParam& refSPH,
	long long stream)
{
	if (sph.size == 0) return;

	ApplyDFSPH_kernel << < mcuda::util::DivUp(sph.size, nBlockSize), nBlockSize, 0, reinterpret_cast<hipStream_t>(stream) >> > (
		sph.pPosition,
		sph.pRadius,
		sph.pPressure,
		sph.pForce,
		sph.size,
		pRefSPHMaterial,
		refSPH.pPosition,
		refSPH.pRadius,
		refSPH.pPressure,
		nei.pID,
		nei.pIdx);
	CUDA_CHECK(hipPeekAtLastError());
}

void mps::kernel::SPH::ApplyDFSPHSub(
	const mps::SPHParam& sph,
	const mps::NeiParam& nei,
	const mps::BoundaryParticleParam& boundaryParticle,
	long long stream)
{
	if (sph.size == 0) return;

	ApplyDFSPH_kernel << < mcuda::util::DivUp(sph.size, nBlockSize), nBlockSize, 0, reinterpret_cast<hipStream_t>(stream) >> > (
		sph.pPosition,
		sph.pRadius,
		sph.pPressure,
		sph.pForce,
		sph.size,
		boundaryParticle.pPosition,
		boundaryParticle.pRadius,
		boundaryParticle.pVolume,
		nei.pID,
		nei.pIdx);
	CUDA_CHECK(hipPeekAtLastError());
}

void mps::kernel::SPH::ApplyDFSPHFinal(
	const mps::SPHParam& sph)
{
	if (sph.size == 0) return;

	ApplyDFSPHFinal_kernel << < mcuda::util::DivUp(sph.size, nFullBlockSize), nFullBlockSize >> > (
		sph.pMass,
		sph.pForce,
		sph.size);
	CUDA_CHECK(hipPeekAtLastError());
}

void mps::kernel::SPH::ComputeDFSPHConstantDensity(
	const mps::PhysicsParam& physParam,
	const mps::SPHMaterialParam& sphMaterial,
	const mps::SPHParam& sph,
	const mps::MeshMaterialParam& boundaryParticleMaterial,
	const mps::BoundaryParticleParam& boundaryParticle,
	const mps::NeiParam& neiSPH2SPH,
	const mps::NeiParam& neiSPH2BoundaryParticle)
{
	if (sph.size == 0) return;

#if SPH_TIMER_PRINT
	hipDeviceSynchronize();
	MTimer::Start("ComputeDFSPHConstantDensity");
#endif

	hipDeviceSynchronize();
	MTimer::Start("CreateStream");
	std::vector<hipStream_t> aStream(3);
	hipStreamCreateWithFlags(&aStream[0], hipStreamNonBlocking);
	hipStreamCreateWithFlags(&aStream[1], hipStreamNonBlocking);
	hipStreamCreateWithFlags(&aStream[2], hipStreamNonBlocking);
	hipDeviceSynchronize();
	MTimer::End("CreateStream");

	thrust::device_vector<REAL> d_error(1);
	thrust::host_vector<REAL> h_error(1);

	uint32_t l = 1u;
	while (l <= 100u)
	{
		d_error.front() = static_cast<REAL>(0.0);

		hipDeviceSynchronize();
		MTimer::Start("fill");
		thrust::fill(
			thrust::device_pointer_cast(sph.pTempReal),
			thrust::device_pointer_cast(sph.pTempReal + sph.size),
			static_cast<REAL>(0.0));
		hipDeviceSynchronize();
		MTimer::End("fill");

		hipDeviceSynchronize();
		MTimer::Start("ComputeDensityDeltaSub");
		ComputeDensityDeltaSub(sphMaterial, sph, neiSPH2SPH, reinterpret_cast<long long>(aStream[0]));
		ComputeDensityDeltaSub(sph, neiSPH2BoundaryParticle, boundaryParticle, reinterpret_cast<long long>(aStream[1]));
		hipDeviceSynchronize();

		/*ComputeDensityDeltaSub(sphMaterial, sph, neiSPH2SPH);
		ComputeDensityDeltaSub(sph, neiSPH2BoundaryParticle, boundaryParticle);*/
		hipDeviceSynchronize();
		MTimer::End("ComputeDensityDeltaSub");

		hipDeviceSynchronize();
		MTimer::Start("ComputeDFSPHConstantDensitySub");
		ComputeDFSPHConstantDensitySub(physParam, sphMaterial, sph, thrust::raw_pointer_cast(d_error.data()));
		hipDeviceSynchronize();
		MTimer::End("ComputeDFSPHConstantDensitySub");

		hipDeviceSynchronize();
		MTimer::Start("h_error");
		h_error = d_error;
		h_error.front() /= static_cast<REAL>(sph.size);
		if (h_error.front() < 1.0e-4 && l > 2u) break;

	#if SPH_DEBUG_PRINT
		std::stringstream ss;
		ss << "Pressure Error " << l << " : " << h_error.front() << std::endl;
		OutputDebugStringA(ss.str().c_str());
	#endif
		hipDeviceSynchronize();
		MTimer::End("h_error");


		hipDeviceSynchronize();
		MTimer::Start("fill");
		thrust::fill(thrust::device_pointer_cast(sph.pForce), thrust::device_pointer_cast(sph.pForce + sph.size), REAL3{ 0.0 });
		hipDeviceSynchronize();
		MTimer::End("fill");

		hipDeviceSynchronize();
		MTimer::Start("ApplyDFSPHSub");
		ApplyDFSPHSub(sphMaterial, sph, neiSPH2SPH, reinterpret_cast<long long>(aStream[0]));
		ApplyDFSPHSub(sph, neiSPH2BoundaryParticle, boundaryParticle, reinterpret_cast<long long>(aStream[1]));
		hipDeviceSynchronize();

		/*ApplyDFSPHSub(sphMaterial, sph, neiSPH2SPH);
		ApplyDFSPHSub(sph, neiSPH2BoundaryParticle, boundaryParticle);*/
		hipDeviceSynchronize();
		MTimer::End("ApplyDFSPHSub");

		hipDeviceSynchronize();
		MTimer::Start("ApplyDFSPHFinal");
		ApplyDFSPHFinal(sph);
		mps::kernel::Advect::UpdateVelocity(physParam, sph);
		hipDeviceSynchronize();
		MTimer::End("ApplyDFSPHFinal");
		l++;
	}

	hipDeviceSynchronize();
	MTimer::Start("hipStreamDestroy");
	hipStreamDestroy(aStream[0]);
	hipStreamDestroy(aStream[1]);
	hipStreamDestroy(aStream[2]);
	hipDeviceSynchronize();
	MTimer::End("hipStreamDestroy");

#if SPH_TIMER_PRINT
	hipDeviceSynchronize();
	std::stringstream ss;
	ss << "Loop " << " : " << l;
	MTimer::EndWithMessage("ComputeDFSPHConstantDensity", ss.str());
#endif
}
void mps::kernel::SPH::ComputeDFSPHDivergenceFree(
	const mps::PhysicsParam& physParam,
	const mps::SPHMaterialParam& sphMaterial,
	const mps::SPHParam& sph,
	const mps::MeshMaterialParam& boundaryParticleMaterial,
	const mps::BoundaryParticleParam& boundaryParticle,
	const mps::NeiParam& neiSPH2SPH,
	const mps::NeiParam& neiSPH2BoundaryParticle)
{
	if (sph.size == 0) return;

#if SPH_TIMER_PRINT
	hipDeviceSynchronize();
	MTimer::Start("ComputeDFSPHDivergenceFree");
#endif

	thrust::device_vector<REAL> d_error(1);
	thrust::host_vector<REAL> h_error(1);

	uint32_t l = 1u;
	while (l <= 100u)
	{
		d_error.front() = static_cast<REAL>(0.0);

		thrust::fill(
			thrust::device_pointer_cast(sph.pTempReal),
			thrust::device_pointer_cast(sph.pTempReal + sph.size),
			static_cast<REAL>(0.0));
		ComputeDensityDeltaSub(sphMaterial, sph, neiSPH2SPH);
		ComputeDensityDeltaSub(sph, neiSPH2BoundaryParticle, boundaryParticle);
		ComputeDFSPHDivergenceFreeSub(physParam, sphMaterial, sph, thrust::raw_pointer_cast(d_error.data()));

		h_error = d_error;
		h_error.front() /= static_cast<REAL>(sph.size);
		if (h_error.front() < 1.0e-3 && l > 1u) break;

	#if SPH_DEBUG_PRINT
		std::stringstream ss;
		ss << "DivergenceFree Error " << l << " : " << h_error.front() << std::endl;
		OutputDebugStringA(ss.str().c_str());
	#endif

		thrust::fill(thrust::device_pointer_cast(sph.pForce), thrust::device_pointer_cast(sph.pForce + sph.size), REAL3{ 0.0 });
		ApplyDFSPHSub(sphMaterial, sph, neiSPH2SPH);
		ApplyDFSPHSub(sph, neiSPH2BoundaryParticle, boundaryParticle);
		ApplyDFSPHFinal(sph);
		mps::kernel::Advect::UpdateVelocity(physParam, sph);
		l++;
	}

#if SPH_TIMER_PRINT
	hipDeviceSynchronize();
	std::stringstream ss;
	ss << "Loop " << " : " << l;
	MTimer::EndWithMessage("ComputeDFSPHDivergenceFree", ss.str());
#endif
}