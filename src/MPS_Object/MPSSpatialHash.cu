#include "stdafx.h"
#include "MPSSpatialHash.cuh"
#include <thrust/sort.h>

#include "MPSSPHParam.h"
#include "MPSBoundaryParticleParam.h"

namespace
{
	constexpr auto nBlockSize = 1024u;
}

mps::SpatialHash::SpatialHash() : mps::VirtualTree<SpatialHashParam>{}
{
}

void mps::SpatialHash::SetObjectSize(const size_t size)
{
	m_key.resize(size);
	m_ID.resize(size);

	GetParam().SetObjectSize(size);
	GetParam().SetKeyArray(thrust::raw_pointer_cast(m_key.data()));
	GetParam().SetIDArray(thrust::raw_pointer_cast(m_ID.data()));
}

void mps::SpatialHash::SetCeilSize(const REAL size)
{
	GetParam().SetCeilSize(size);
}

void mps::SpatialHash::SetHashSize(const glm::uvec3& size)
{
	const auto hashCeilSize = size.x * size.y * size.z;
	m_startIdx.resize(hashCeilSize);
	m_endIdx.resize(hashCeilSize);

	GetParam().SetHashSize(size);
	GetParam().SetStartIdxArray(thrust::raw_pointer_cast(m_startIdx.data()));
	GetParam().SetEndIdxArray(thrust::raw_pointer_cast(m_endIdx.data()));
}

void mps::SpatialHash::UpdateHash(const mps::ObjectParam& objParam)
{
	const auto nSize = objParam.GetSize();
	if (nSize == 0) return;

	thrust::fill(m_startIdx.begin(), m_startIdx.end(), 0xffffffff);
	thrust::fill(m_endIdx.begin(), m_endIdx.end(), 0xffffffff);

	InitHash_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> >
		(GetParam(), objParam);
	CUDA_CHECK(hipPeekAtLastError());

	thrust::sort_by_key(m_key.begin(), m_key.end(), m_ID.begin());

	ReorderHash_kernel << <mcuda::util::DivUp(nSize, nBlockSize), nBlockSize, (nBlockSize + 1) * sizeof(uint32_t) >> >
		(GetParam());
	CUDA_CHECK(hipPeekAtLastError());
}

void mps::SpatialHash::ZSort(mps::ObjectParam& objParam)
{
	const auto nSize = objParam.GetSize();
	if (nSize == 0) return;

	thrust::fill(m_startIdx.begin(), m_startIdx.end(), 0xffffffff);
	thrust::fill(m_endIdx.begin(), m_endIdx.end(), 0xffffffff);

	InitHashZIndex_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> >
		(GetParam(), objParam);
	CUDA_CHECK(hipPeekAtLastError());

	thrust::sort_by_key(m_key.begin(), m_key.end(), thrust::make_zip_iterator(
		thrust::device_pointer_cast(objParam.GetPosArray()),
		thrust::device_pointer_cast(objParam.GetMassArray()),
		thrust::device_pointer_cast(objParam.GetVelocityArray()),
		thrust::device_pointer_cast(objParam.GetColorArray())));
}

void mps::SpatialHash::ZSort(mps::SPHParam& sphParam)
{
	const auto nSize = sphParam.GetSize();
	if (nSize == 0) return;

	thrust::fill(m_startIdx.begin(), m_startIdx.end(), 0xffffffff);
	thrust::fill(m_endIdx.begin(), m_endIdx.end(), 0xffffffff);

	InitHashZIndex_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> >
		(GetParam(), sphParam);
	CUDA_CHECK(hipPeekAtLastError());

	thrust::sort_by_key(m_key.begin(), m_key.end(), thrust::make_zip_iterator(
		thrust::device_pointer_cast(sphParam.GetPosArray()),
		thrust::device_pointer_cast(sphParam.GetMassArray()),
		thrust::device_pointer_cast(sphParam.GetVelocityArray()),
		thrust::device_pointer_cast(sphParam.GetColorArray()),
		thrust::device_pointer_cast(sphParam.GetRadiusArray())));
}

void mps::SpatialHash::ZSort(mps::BoundaryParticleParam& boundaryParticleParam)
{
	const auto nSize = boundaryParticleParam.GetSize();
	if (nSize == 0) return;

	InitHashZIndex_kernel << < mcuda::util::DivUp(nSize, nBlockSize), nBlockSize >> >
		(GetParam(), boundaryParticleParam);
	CUDA_CHECK(hipPeekAtLastError());

	thrust::sort_by_key(m_key.begin(), m_key.end(), thrust::make_zip_iterator(
		thrust::device_pointer_cast(boundaryParticleParam.GetPosArray()),
		thrust::device_pointer_cast(boundaryParticleParam.GetMassArray()),
		thrust::device_pointer_cast(boundaryParticleParam.GetVelocityArray()),
		thrust::device_pointer_cast(boundaryParticleParam.GetColorArray()),
		thrust::device_pointer_cast(boundaryParticleParam.GetRadiusArray()),
		thrust::device_pointer_cast(boundaryParticleParam.GetFaceIDArray()),
		thrust::device_pointer_cast(boundaryParticleParam.GetBCCArray())));
}
